
#include "macrodef.h"
#include "functional.h"
#include "ntensor.h"


int main(){
    GEN_VAR_WITH_VALUE(double, a_value, 1.0, (unsigned)2<<24);
    GEN_VAR_WITH_VALUE(double, b_value, 2.0, (unsigned)2<<24);
    GEN_VAR_WITH_VALUE(double, c_value, 1.5, (unsigned)2<<24);
    GEN_VAR_WITH_VALUE(double, root_grad, 1.0, (unsigned)2<<24);
    int id=0;
    #ifdef DEVICE_GPU
        hipGetDevice(&id);
    #endif
    std::cout<<"CUDA ID: "<<id<<std::endl;
    auto start = std::chrono::high_resolution_clock::now();
    
    Ntensor X(a_value,{1,(unsigned)2<<24},NULL,NULL,std::string(), NULL), 
        W(b_value,{1,(unsigned)2<<24},NULL,NULL,std::string(),NULL), 
        B(c_value,{1,(unsigned)2<<24},NULL,NULL,std::string(),NULL),
        C(c_value,{1,(unsigned)2<<24},NULL,NULL,std::string(),NULL),
        Y(c_value,{1,(unsigned)2<<24},NULL,NULL,std::string(),NULL),
        P_Y(c_value,{1,(unsigned)2<<24},NULL,NULL,std::string(),NULL),
        L(c_value,{1,(unsigned)2<<24},NULL,NULL,std::string(),NULL)
        ;
    C=W*X;
    Y=sigmoid(C)+B;
    L=P_Y-Y;
    L.backward(root_grad);
    
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    std::cout << "Elapsed time: " << duration << " ms" << std::endl;
    return 0;
}
